#include <hip/hip_runtime.h>
#include <iostream>

#ifndef NDEBUG
#error "NDEBUG not defined, this is a Meson bug"
#endif

int cuda_devices(void) {
    int result = 0;
    hipGetDeviceCount(&result);
    return result;
}


int main(void) {
    int n = cuda_devices();
    if (n == 0) {
        std::cout << "No Cuda hardware found. Exiting.\n";
        return 0;
    }

    std::cout << "Found " << n << "Cuda devices.\n";
    return 0;
}
