#include <stdio.h>
#include <hip/hip_runtime.h>
#include "shared/kernels.h"


int main(void) {
    int cuda_devices = 0;
    hipGetDeviceCount(&cuda_devices);
    if(cuda_devices == 0) {
        printf("No Cuda hardware found. Exiting.\n");
        return 0;
    }

    if(run_tests() != 0){
        printf("CUDA tests failed! Exiting.\n");
        return 0;
    }

    return 0;
}
