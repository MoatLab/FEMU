#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "kernels.h"


TAG_HIDDEN __global__ void kernel (void){
}

TAG_PUBLIC int run_tests(void) {
  kernel<<<1,1>>>();

  return (int)hipDeviceSynchronize();
}
