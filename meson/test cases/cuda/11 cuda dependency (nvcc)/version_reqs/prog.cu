#include <hip/hip_runtime.h>
#include <iostream>

int cuda_devices(void) {
    int result = 0;
    hipGetDeviceCount(&result);
    return result;
}

int main(void) {
    std::cout << "Compiled against CUDA version: " << CUDART_VERSION << "\n";

    int runtime_version = 0;
    switch (hipError_t r = hipRuntimeGetVersion(&runtime_version)) {
        case hipSuccess:
            std::cout << "CUDA runtime version: " << runtime_version << "\n";
            break;
        case hipErrorNoDevice:
            std::cout << "No CUDA hardware found. Exiting.\n";
            return 0;
        default:
            std::cout << "Couldn't obtain CUDA runtime version (error " << r << "). Exiting.\n";
            return -1;
    }

    int n = cuda_devices();
    std::cout << "Found " << n << " CUDA devices.\n";
    return 0;
}
