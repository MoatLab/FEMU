
#include <hip/hip_runtime.h>
#include <iostream>

int main(void) {
    int cuda_devices = 0;
    std::cout << "CUDA version: " << CUDART_VERSION << "\n";
    hipGetDeviceCount(&cuda_devices);
    if(cuda_devices == 0) {
        std::cout << "No Cuda hardware found. Exiting.\n";
        return 0;
    }
    std::cout << "This computer has " << cuda_devices << " Cuda device(s).\n";
    hipDeviceProp_t props;
    hipGetDeviceProperties(&props, 0);
    std::cout << "Properties of device 0.\n\n";

    std::cout << "  Name:            " << props.name << "\n";
    std::cout << "  Global memory:   " << props.totalGlobalMem << "\n";
    std::cout << "  Shared memory:   " << props.sharedMemPerBlock << "\n";
    std::cout << "  Constant memory: " << props.totalConstMem << "\n";
    std::cout << "  Block registers: " << props.regsPerBlock << "\n";

    std::cout << "  Warp size:         " << props.warpSize << "\n";
    std::cout << "  Threads per block: " << props.maxThreadsPerBlock << "\n";
    std::cout << "  Max block dimensions: [ " << props.maxThreadsDim[0] << ", " << props.maxThreadsDim[1]  << ", " << props.maxThreadsDim[2] << " ]" << "\n";
    std::cout << "  Max grid dimensions:  [ " << props.maxGridSize[0] << ", " << props.maxGridSize[1]  << ", " << props.maxGridSize[2] << " ]" << "\n";
    std::cout << "\n";

    return 0;
}
